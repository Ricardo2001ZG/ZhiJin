
#include <hip/hip_runtime.h>
//
// Simple cuda test code
//

__global__ void increment(int *a) 
{
	a[threadIdx.x] += 1; // b[threadIdx.x];
}

int main()
{
	const int dataSize = 16; 
	int a[dataSize] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10,11,12,13,14,15,16};

	// allocate work buffer
	int *ad;
	const int bufferSize = dataSize * sizeof(int);
	hipMalloc( (void**)&ad, bufferSize ); 

	// copy input to work buffer
	hipMemcpy( ad, a, bufferSize, hipMemcpyHostToDevice ); 

	// do increments
	dim3 dimBlock( dataSize, 1 );
	dim3 dimGrid( 1, 1 );
	increment<<<dimGrid, dimBlock>>>(ad);

	// copy back result
	hipMemcpy( a, ad, bufferSize, hipMemcpyDeviceToHost );

	// cleanup
	hipFree( ad );
	
	return 0;
}